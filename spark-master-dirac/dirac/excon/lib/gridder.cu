#include "hip/hip_runtime.h"
/*
 *
   Copyright (C) 2013 Sarod Yatawatta <sarod@users.sf.net>  
 This program is free software; you can redistribute it and/or modify
 it under the terms of the GNU General Public License as published by
 the Free Software Foundation; either version 2 of the License, or
 (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software
 Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
 $Id$
*/

#include "gridder.h"

//declare texture reference
texture<hipFloatComplex,hipTextureType3D,hipReadModeElementType> texreference;
//texture<hipFloatComplex,hipTextureType2DLayered,hipReadModeElementType> texreference;


/* search for val in array list of size nx1 */
__global__ static void cuda_search(int *out, float *list, float val, float lambda, int n)
{
    int start = threadIdx.x + blockIdx.x * blockDim.x;
    /* take abs(w) in lambda for wavelength conversion */
    float wl=fabs(val);
    for (int idx = start; idx < n; idx += gridDim.x * blockDim.x) {
        if (list[idx] >= wl) return;
        float next = list[idx + 1];
        if (idx == n-1 || next >= wl) {
            *out =( next == wl ? (idx + 1) : idx);
            return;
        }
     }
}


__device__ static int cuda_ifftshift_index(int N, int x) {
 if (N%2) { /* odd */
  return((x+N/2)%N);
 } else { /* even */
  return((x+N/2+1)%N);
 }
}

/* reset memory to Nx x Ny to eliminate points off grid */
__global__ void cuda_resetmemory(unsigned long int *outoff, int Mx, int Nx, int Ny) {
    int ui = threadIdx.x + blockIdx.x * blockDim.x;
    int vi = threadIdx.y + blockIdx.y * blockDim.y;
    size_t oidx = 2*(vi+ui*Mx);
    if (ui<Mx && vi<Mx) {
     outoff[oidx]=Nx*Ny;
     outoff[oidx+1]=Nx*Ny;
    }
    __syncthreads();
}


/* total support is Mx pixels, so total pixels is Mx x Mx 
 since we also grid the -ve u,v,w point, total data size is 2 Mx x Mx */
__global__ void cuda_griddata(hipFloatComplex* outputimg, hipFloatComplex *outputpsf, unsigned long int *outoff, int Mx, float u, float v, float w, float wt, hipFloatComplex xx, hipFloatComplex yy, hipFloatComplex xy, hipFloatComplex yx, float maxW, float expW, float lambda, float uvscale, float deltaU, int Nx, int Ny, int Nw, int Np, int Nz)
{

    /* note: v axis is negative */
    /* all position calculations are done using double precision */
    __shared__ float zl,zh;
    __shared__ float wh,wl,wa;
    __shared__ double uf,vf,up,vp;
    __shared__ int signofw;
    __shared__ double pix0;
    __shared__ hipFloatComplex sI;
    __shared__ double NxNp;
    if (threadIdx.x == 0) {
      pix0=-sqrt(0.8); //trueimg/zeropaddedimg?? old -1
      sI=hipCmulf(hipCaddf(xx,yy),make_hipFloatComplex(0.5f,0.0f));
      /* u,v: corresponding pixel value */
      uf=double(u)*double(uvscale); /* in [-0.5,0.5] */
      vf=double(v)*double(uvscale); /* in [-0.5,0.5] */
      /* nearest central pixel */
      up=rint(uf*double(Nx)+0.5*double(Nx))-1.0; /* in [0,Nx-1] */
      vp=rint(-vf*double(Ny)+0.5*double(Ny))-1.0; /* in [0,Ny-1] */
 
      float deltaz=fdividef(0.1f,(float)Nw); /* 0.1/Nw */
      wa=fdividef(fabsf(w),maxW);
      /* w in normalized coords [0,1], with sqrt() spacing */
      //float z=sqrtf(wa); /* to make general, use power, exponent __powf(x,y) */
      float z=powf(wa,expW);
      zl=z-deltaz;
      zl=(zl<0.0f?0.0f:zl);
      zh=z+deltaz;
      zh=(zh>1.0f?1.0f:zh);
      //wl=zl*zl;
      //wh=zh*zh;
      wl=powf(zl,1.0f/expW);
      wh=powf(zh,1.0f/expW);
      /* sign of the w term */
      signofw=signbit(w);
      NxNp=1.0/(double(Nx*Np)*double(deltaU));
    }
    __syncthreads();
    /* ui,vi: in [0,Mx-1] */
    int ui = threadIdx.x + blockIdx.x * blockDim.x;
    int vi = threadIdx.y + blockIdx.y * blockDim.y;
    /* absolute pixel value for this thread in [0,Nx-1], [0,Ny-1] */
    int x0 = int(up)-(Mx>>1)+ui;
    int y0 = int(vp)-(Mx>>1)+vi;
    int x1 = cuda_ifftshift_index(Nx,x0);
    int y1 = cuda_ifftshift_index(Ny,y0);

    /* normalized distance of this pixel from central pixel : in images pixels of [0,Nx-1] => scale to [0,Nz-1] and note that Np pixels map to [0,1]  */
    float x = float(((uf+0.5)*double(Nx)-double(x0)+pix0)*NxNp+0.5);
    float y = float(((-vf+0.5)*double(Ny)-double(y0)+pix0)*NxNp+0.5);

    size_t oidx = 2*(vi+ui*Mx);
    /* apart from checking the support, also throw out -ve pixels */
    if (ui<Mx && vi<Mx && x0>=0 && x0<Nx && y0>=0 && y0<Ny) {
     hipFloatComplex psfl=tex3D(texreference, x, y, zl);
     hipFloatComplex psfh=tex3D(texreference, x, y, zh);
     hipFloatComplex psf;
     psf.x=fdividef(psfl.x*(wh-wa)+psfh.x*(wa-wl),wh-wl);
     psf.y=fdividef(psfl.y*(wh-wa)+psfh.y*(wa-wl),wh-wl);
     /* multiply pswf with inverse sigma */
     psf=hipCmulf(make_hipFloatComplex(wt,0.0f),psf);
#ifdef ONE_GPU
     //printf("Np x delu=%f, u,v=%f,%f, scale=%f, deltau=%f, uf,vf=%f,%f up,vp=%f,%f ui,vi=(%d,%d) x0,y0=(%d,%d) x1,y1=(%d,%d) (x,y,z)=%f,%f,%f pswf=%f,%f\n",float(Np)*deltaU,u,v,uvscale,deltaU,uf,vf,up,vp,ui,vi,x0,y0,x1,y1,x,y,z,psf.x,psf.y);
     //printf("%f %f %f\n",x,y,z);
#endif
     if (signofw) { /* w is negative */
        psf=hipConjf(psf);
     }
     outputimg[oidx]=hipCmulf(sI,psf);
     outputpsf[oidx]=psf;
     outoff[oidx]=x1*Ny+y1;
    }
    __syncthreads();

    /* now handle -ve u,v,w point */
    if (threadIdx.x == 0) {
      /* flip sign of u,v */
      uf=-uf; /* in [-0.5,0.5] */
      vf=-vf; /* in [-0.5,0.5] */
      /* nearest central pixel */
      up=-rint(-(uf*double(Nx)+0.5*double(Nx)))-1.0; /* in [0,Nx-1] */
      vp=-rint(-(-vf*double(Ny)+0.5*double(Ny)))-1.0; /* in [0,Ny-1] */
    }
    __syncthreads();

    /* absolute pixel value for this thread in [0,Nx-1], [0,Ny-1] */
    x0 = int(up)-(Mx>>1)+ui;
    y0 = int(vp)-(Mx>>1)+vi;
    x1 = cuda_ifftshift_index(Nx,x0);
    y1 = cuda_ifftshift_index(Ny,y0);

    x = float(((uf+0.5)*double(Nx)-double(x0)+pix0)*NxNp+0.5);
    y = float(((-vf+0.5)*double(Ny)-double(y0)+pix0)*NxNp+0.5);

    oidx = 2*(vi+ui*Mx)+1;
    /* apart from checking the support, also throw out -ve pixels */
    if (ui<Mx && vi<Mx && x0>=0 && x0<Nx && y0>=0 && y0<Ny) {
     hipFloatComplex psfl=tex3D(texreference, x, y, zl);
     hipFloatComplex psfh=tex3D(texreference, x, y, zh);
     hipFloatComplex psf;
     psf.x=fdividef(psfl.x*(wh-wa)+psfh.x*(wa-wl),wh-wl);
     psf.y=fdividef(psfl.y*(wh-wa)+psfh.y*(wa-wl),wh-wl);
     /* multiply pswf with inverse sigma */
     psf=hipCmulf(make_hipFloatComplex(wt,0.0f),psf);
#ifdef ONE_GPU
     //printf("Np x delu=%f, u,v=%f,%f, scale=%f, deltau=%f, uf,vf=%f,%f up,vp=%f,%f ui,vi=(%d,%d) x0,y0=(%d,%d) x1,y1=(%d,%d) (x,y,z)=%f,%f,%f pswf=%f,%f\n",float(Np)*deltaU,u,v,uvscale,deltaU,uf,vf,up,vp,ui,vi,x0,y0,x1,y1,x,y,z,psf.x,psf.y);
     //printf("%f %f %f\n",x,y,z);
#endif
     //hipFloatComplex psf=tex2DLayered(texreference, x, y, z);
     if (!signofw) { /* -w is negative */
        psf=hipConjf(psf);
     }
     outputimg[oidx]=hipCmulf(hipConjf(sI),psf);
     outputpsf[oidx]=psf;
     outoff[oidx]=x1*Ny+y1;
    }
    __syncthreads();
}

/* total support is Mx pixels, so total pixels is Mx x Mx 
 since we also grid the -ve u,v,w point, total data size is 2 Mx x Mx */
__global__ void cuda_griddata_iquv(hipFloatComplex* outputimg, hipFloatComplex *outputpsf, unsigned long int *outoff, hipFloatComplex* outputimgQ, hipFloatComplex* outputimgU, hipFloatComplex* outputimgV, int Mx, float u, float v, float w, float wt, hipFloatComplex xx, hipFloatComplex yy, hipFloatComplex xy, hipFloatComplex yx, float maxW, float expW, float lambda, float uvscale, float deltaU, int Nx, int Ny, int Nw, int Np, int Nz)
{

    /* note: v axis is negative */
    __shared__ float zl,zh;
    __shared__ float wh,wl,wa;
    __shared__ double uf,vf,up,vp;
    __shared__ int signofw;
    __shared__ float pix0;
    __shared__ hipFloatComplex sI,sQ,sU,sV;
    __shared__ double NxNp;
    if (threadIdx.x == 0) {
      pix0=-sqrt(0.8); //trueimg/zeropaddedimg?? old -1
      sI=hipCmulf(hipCaddf(xx,yy),make_hipFloatComplex(0.5f,0.0f));
      /* Q =(XX-YY)/2 U=(XY+YX)/2 V=imag(YX-XY)/2 */
      sQ=hipCmulf(hipCsubf(xx,yy),make_hipFloatComplex(0.5f,0.0f));
      sU=hipCmulf(hipCaddf(xy,yx),make_hipFloatComplex(0.5f,0.0f));
      sV=hipCmulf(hipCsubf(yx,xy),make_hipFloatComplex(0.0f,0.5f));
      /* u,v: corresponding pixel value */
      uf=double(u)*double(uvscale); /* in [-0.5,0.5] */
      vf=double(v)*double(uvscale); /* in [-0.5,0.5] */
      /* nearest central pixel pixel */
      up=rint(uf*double(Nx)+0.5*double(Nx))-1.0; /* in [0,Nx-1] */
      vp=rint(-vf*double(Ny)+0.5*double(Ny))-1.0; /* in [0,Ny-1] */
 
      float deltaz=fdividef(0.1f,(float)Nw); /* 0.1/Nw */
      wa=fdividef(fabsf(w),maxW);
      /* w in normalized coords [0,1], with w^expW spacing */
      //float z=sqrtf(wa);
      float z=powf(wa,expW);
      zl=z-deltaz;
      zl=(zl<0.0f?0.0f:zl);
      zh=z+deltaz;
      zh=(zh>1.0f?1.0f:zh);
      //wl=zl*zl;
      //wh=zh*zh;
      wl=powf(zl,1.0f/expW);
      wh=powf(zh,1.0f/expW);
      /* sign of the w term */
      signofw=signbit(w);
      NxNp=1.0/(double(Nx*Np)*double(deltaU));
    }
    __syncthreads();
    /* ui,vi: in [0,Mx-1] */
    int ui = threadIdx.x + blockIdx.x * blockDim.x;
    int vi = threadIdx.y + blockIdx.y * blockDim.y;
    /* absolute pixel value for this thread in [0,Nx-1], [0,Ny-1] */
    int x0 = int(up)-(Mx>>1)+ui;
    int y0 = int(vp)-(Mx>>1)+vi;
    int x1 = cuda_ifftshift_index(Nx,x0);
    int y1 = cuda_ifftshift_index(Ny,y0);

    /* normalized distance of this pixel from central pixel : in images pixels of [0,Nx-1] => scale to [0,Nz-1] and note that Np pixels map to [0,1]  */
    float x = float(((uf+0.5)*double(Nx)-double(x0)+pix0)*NxNp+0.5);
    float y = float(((-vf+0.5)*double(Ny)-double(y0)+pix0)*NxNp+0.5);

    size_t oidx = 2*(vi+ui*Mx);
    /* apart from checking the support, also throw out -ve pixels */
    if (ui<Mx && vi<Mx && x0>=0 && x0<Nx && y0>=0 && y0<Ny) {
     hipFloatComplex psfl=tex3D(texreference, x, y, zl);
     hipFloatComplex psfh=tex3D(texreference, x, y, zh);
     hipFloatComplex psf;
     psf.x=fdividef(psfl.x*(wh-wa)+psfh.x*(wa-wl),wh-wl);
     psf.y=fdividef(psfl.y*(wh-wa)+psfh.y*(wa-wl),wh-wl);
     /* multiply pswf with inverse sigma */
     psf=hipCmulf(make_hipFloatComplex(wt,0.0f),psf);
#ifdef ONE_GPU
     //printf("Np x delu=%f, u,v=%f,%f, scale=%f, deltau=%f, uf,vf=%f,%f up,vp=%f,%f ui,vi=(%d,%d) x0,y0=(%d,%d) x1,y1=(%d,%d) (x,y,z)=%f,%f,%f pswf=%f,%f\n",float(Np)*deltaU,u,v,uvscale,deltaU,uf,vf,up,vp,ui,vi,x0,y0,x1,y1,x,y,z,psf.x,psf.y);
     //printf("%f %f %f\n",x,y,z);
#endif
     if (signofw) { /* w is negative */
        psf=hipConjf(psf);
     }
     outputpsf[oidx]=psf;
     outputimg[oidx]=hipCmulf(sI,psf);
     outoff[oidx]=x1*Ny+y1;
     outputimgQ[oidx]=hipCmulf(sQ,psf);
     outputimgU[oidx]=hipCmulf(sU,psf);
     outputimgV[oidx]=hipCmulf(sV,psf);
    }
    __syncthreads();

    /* now handle -ve u,v,w point */
    if (threadIdx.x == 0) {
      /* flip sign of u,v */
      uf=-uf; /* in [-0.5,0.5] */
      vf=-vf; /* in [-0.5,0.5] */
      /* nearest central pixel pixel */
      up=-rint(-(uf*double(Nx)+0.5f*double(Nx)))-1.0; /* in [0,Nx-1] */
      vp=-rint(-(-vf*double(Ny)+0.5f*double(Ny)))-1.0; /* in [0,Ny-1] */
    }
    __syncthreads();

    /* absolute pixel value for this thread in [0,Nx-1], [0,Ny-1] */
    x0 = int(up)-(Mx>>1)+ui;
    y0 = int(vp)-(Mx>>1)+vi;
    x1 = cuda_ifftshift_index(Nx,x0);
    y1 = cuda_ifftshift_index(Ny,y0);

    x = float(((uf+0.5)*double(Nx)-double(x0)+pix0)*NxNp+0.5);
    y = float(((-vf+0.5)*double(Ny)-double(y0)+pix0)*NxNp+0.5);

    oidx = 2*(vi+ui*Mx)+1;
    /* apart from checking the support, also throw out -ve pixels */
    if (ui<Mx && vi<Mx && x0>=0 && x0<Nx && y0>=0 && y0<Ny) {
     hipFloatComplex psfl=tex3D(texreference, x, y, zl);
     hipFloatComplex psfh=tex3D(texreference, x, y, zh);
     hipFloatComplex psf;
     psf.x=fdividef(psfl.x*(wh-wa)+psfh.x*(wa-wl),wh-wl);
     psf.y=fdividef(psfl.y*(wh-wa)+psfh.y*(wa-wl),wh-wl);
     /* multiply pswf with inverse sigma */
     psf=hipCmulf(make_hipFloatComplex(wt,0.0f),psf);
#ifdef ONE_GPU
     //printf("Np x delu=%f, u,v=%f,%f, scale=%f, deltau=%f, uf,vf=%f,%f up,vp=%f,%f ui,vi=(%d,%d) x0,y0=(%d,%d) x1,y1=(%d,%d) (x,y,z)=%f,%f,%f pswf=%f,%f\n",float(Np)*deltaU,u,v,uvscale,deltaU,uf,vf,up,vp,ui,vi,x0,y0,x1,y1,x,y,z,psf.x,psf.y);
     //printf("%f %f %f\n",x,y,z);
#endif
     //hipFloatComplex psf=tex2DLayered(texreference, x, y, z);
     if (!signofw) { /* -w is negative */
        psf=hipConjf(psf);
     }
     outputimg[oidx]=hipCmulf(hipConjf(sI),psf);
     outputpsf[oidx]=psf;
     outoff[oidx]=x1*Ny+y1;
     outputimgQ[oidx]=hipCmulf(hipConjf(sQ),psf);
     outputimgU[oidx]=hipCmulf(hipConjf(sU),psf);
     outputimgV[oidx]=hipCmulf(hipConjf(sV),psf);
    }
    __syncthreads();
}


#ifdef ONE_GPU
__global__ void 
cuda_printtexture(int Nx,int Ny,int Nw) {
 /* only one thread does work */
 if((threadIdx.x==0) && (blockIdx.x==0) ) {
  printf("Nx=%d Ny=%d Nw=%d\n",Nx,Ny,Nw);
  float delx=1.0f/float(Nx);
  float dely=1.0f/float(Ny);
  float delz=1.0f/float(Nw);
  __syncthreads();
  for (int ci=0; ci<Nx; ci++) {
   for (int cj=0; cj<Ny; cj++) {
    for (int ck=0; ck<Nw; ck++) {
     float x=float(ci)*delx;
     float y=float(cj)*dely;
     float z=float(ck)*delz;
     hipFloatComplex pswf=tex3D(texreference, y, x, 0.1f);
     //hipFloatComplex pswf=tex2DLayered(texreference, x, y, z);
     printf("%f %f %f %f %f\n",x,y,z,pswf.x,pswf.y);
     __syncthreads();
    }  
   }
  }
 }
}
#endif

extern "C" {

static void
checkCudaError(hipError_t err, const char *file, int line)
{
    if(!err)
        return;
    fprintf(stderr,"GPU (CUDA): %s %s %d\n", hipGetErrorString(err),file,line);
    exit(EXIT_FAILURE);
}

#ifdef ONE_GPU
static void
debug_show_texture(int card,int Nx,int Ny,int Nw) {
 printf("card=%d, Nx=%d, Ny=%d, Nw=%d\n",card,Nx,Ny,Nw);
 hipSetDevice(card);
 printf("==================\n");
 cuda_printtexture<<<1, 1>>>(Nx,Ny,Nw);
 printf("==================\n");
}
#endif

/* function to write all buffers to output */
/* tid: 0,1,2,3 , select different ordering */
static void
write_buffers_to_output(int bfilled,int Nx,int Ny,pthread_mutex_t *writelock_img,pthread_mutex_t *writelock_psf,float *uvgrid,float *psfgrid,unsigned long int *hostoff,hipFloatComplex *hostimg,hipFloatComplex *hostpsf,int tid) {
 int ci;
 unsigned long int NN=Nx*Ny;
 if (tid%2==0) {
     pthread_mutex_lock(writelock_img);
     for (ci=0; ci<bfilled; ci++) { /* write real,imag parts separately */
      if (hostoff[ci]<NN) { /* FIXME: why need to check this? */
       uvgrid[2*hostoff[ci]]+=hostimg[ci].x;
       uvgrid[2*hostoff[ci]+1]+=hostimg[ci].y;
      }
     }
     pthread_mutex_unlock(writelock_img);
     pthread_mutex_lock(writelock_psf);
     for (ci=0; ci<bfilled; ci++) {
      if (hostoff[ci]<NN) {
       psfgrid[2*hostoff[ci]]+=hostpsf[ci].x;
       psfgrid[2*hostoff[ci]+1]+=hostpsf[ci].y;
      }
     }
     pthread_mutex_unlock(writelock_psf);
 } else if (tid%2==1) {
     pthread_mutex_lock(writelock_psf);
     for (ci=0; ci<bfilled; ci++) {
      if (hostoff[ci]<NN) {
       psfgrid[2*hostoff[ci]]+=hostpsf[ci].x;
       psfgrid[2*hostoff[ci]+1]+=hostpsf[ci].y;
      }
     }
     pthread_mutex_unlock(writelock_psf);
     pthread_mutex_lock(writelock_img);
     for (ci=0; ci<bfilled; ci++) {
      if (hostoff[ci]<NN) {
       uvgrid[2*hostoff[ci]]+=hostimg[ci].x;
       uvgrid[2*hostoff[ci]+1]+=hostimg[ci].y;
      }
     }
     pthread_mutex_unlock(writelock_img);
 } 
}

/* function to write all buffers to output */
/* tid: 0,1,2,3 , select different ordering */
/* not PSF is calculated only for I, so, length/4 for PSF */
static void
write_buffers_to_output_iquv(int bfilled,int Nx,int Ny,pthread_mutex_t *writelock_img,pthread_mutex_t *writelock_psf,float *uvgrid,float *psfgrid,unsigned long int *hostoff,hipFloatComplex *hostimg,hipFloatComplex *hostpsf, 
hipFloatComplex *hostimgQ, hipFloatComplex *hostimgU,hipFloatComplex *hostimgV,
int tid) {
 int ci;
 unsigned long int NN=Nx*Ny;
 if (tid%2==0) {
     pthread_mutex_lock(writelock_img);
     for (ci=0; ci<bfilled; ci++) { /* write real,imag parts separately */
      if (hostoff[ci]<NN) { /* FIXME: why need to check this? */
       uvgrid[2*hostoff[ci]]+=hostimg[ci].x;
       uvgrid[2*hostoff[ci]+1]+=hostimg[ci].y;
       uvgrid[2*(hostoff[ci]+NN)]+=hostimgQ[ci].x;
       uvgrid[2*(hostoff[ci]+NN)+1]+=hostimgQ[ci].y;
      }
     }
     pthread_mutex_unlock(writelock_img);
     pthread_mutex_lock(writelock_psf);
     for (ci=0; ci<bfilled; ci++) {
      if (hostoff[ci]<NN) {
       psfgrid[2*hostoff[ci]]+=hostpsf[ci].x;
       psfgrid[2*hostoff[ci]+1]+=hostpsf[ci].y;
       /* write U,V also here to divide work evenly */
       uvgrid[2*(hostoff[ci]+2*NN)]+=hostimgU[ci].x;
       uvgrid[2*(hostoff[ci]+2*NN)+1]+=hostimgU[ci].y;
       uvgrid[2*(hostoff[ci]+3*NN)]+=hostimgV[ci].x;
       uvgrid[2*(hostoff[ci]+3*NN)+1]+=hostimgV[ci].y;
      }
     }
     pthread_mutex_unlock(writelock_psf);
 } else if (tid%2==1) {
     pthread_mutex_lock(writelock_psf);
     for (ci=0; ci<bfilled; ci++) {
      if (hostoff[ci]<NN) {
       psfgrid[2*hostoff[ci]]+=hostpsf[ci].x;
       psfgrid[2*hostoff[ci]+1]+=hostpsf[ci].y;
       /* write U,V also here to divide work evenly */
       uvgrid[2*(hostoff[ci]+2*NN)]+=hostimgU[ci].x;
       uvgrid[2*(hostoff[ci]+2*NN)+1]+=hostimgU[ci].y;
       uvgrid[2*(hostoff[ci]+3*NN)]+=hostimgV[ci].x;
       uvgrid[2*(hostoff[ci]+3*NN)+1]+=hostimgV[ci].y;
      }
     }
     pthread_mutex_unlock(writelock_psf);
     pthread_mutex_lock(writelock_img);
     for (ci=0; ci<bfilled; ci++) {
      if (hostoff[ci]<NN) {
       uvgrid[2*hostoff[ci]]+=hostimg[ci].x;
       uvgrid[2*hostoff[ci]+1]+=hostimg[ci].y;
       uvgrid[2*(hostoff[ci]+NN)]+=hostimgQ[ci].x;
       uvgrid[2*(hostoff[ci]+NN)+1]+=hostimgQ[ci].y;
      }
     }
     pthread_mutex_unlock(writelock_img);
 } 
}



/* function to set up a GPU, should be called only once */
static void
attach_gpu_to_thread(int card, float *wkernel, int Np, int Nw, float *wparr, float **dwparr, void** carray) {
 hipError_t err;

 hipExtent volumesize;
 hipExtent volumesizeBytes;
 hipChannelFormatDesc channel;
 hipMemcpy3DParms copyparms={0};
 hipArray *cudaarr=0;

 hipSetDevice(card);

 // copy w axis to device
 err=hipMalloc((void**)dwparr, sizeof(float)*(Nw+2));
 checkCudaError(err,__FILE__,__LINE__);
 err=hipMemcpy(*dwparr, (void*)wparr, sizeof(float)*(Nw+2), hipMemcpyHostToDevice);
 checkCudaError(err,__FILE__,__LINE__);

 //set cuda array volume size NOTE: first dimension is in elements, not in
 // bytes as we use hipMalloc3DArray and not hipMalloc
 volumesize=make_hipExtent(Np,Np,Nw);
 volumesizeBytes=make_hipExtent(sizeof(hipFloatComplex)*Np,Np,Nw);

 hipPitchedPtr d_volumeMem;
 err=hipMalloc3D(&d_volumeMem, volumesizeBytes);
 checkCudaError(err,__FILE__,__LINE__);

 err=hipMemcpy(d_volumeMem.ptr, (void*)wkernel, sizeof(hipFloatComplex)*Np*Np*Nw, hipMemcpyHostToDevice);
 checkCudaError(err,__FILE__,__LINE__);

 //create channel to describe data type
 channel=hipCreateChannelDesc<hipFloatComplex>();

 //allocate device memory for cuda array
 //err=hipMalloc3DArray(&cudaarr,&channel,volumesize,hipArrayLayered);
 err=hipMalloc3DArray(&cudaarr,&channel,volumesize);
 checkCudaError(err,__FILE__,__LINE__);

 //set cuda array copy parameters
 copyparms.extent=volumesize;
 copyparms.dstArray=cudaarr;
 copyparms.kind=hipMemcpyDeviceToDevice;
 copyparms.srcPtr=d_volumeMem;
 /* copy data */
 err=hipMemcpy3D(&copyparms);
 checkCudaError(err,__FILE__,__LINE__);
 hipFree(d_volumeMem.ptr);


 //all coordinate axes are in [0,1]
 texreference.normalized=true;
 //set texture filter mode property
 //use hipFilterModePoint or hipFilterModeLinear
 texreference.filterMode=hipFilterModeLinear;
 //set texture address mode property
 //use hipAddressModeClamp or hipAddressModeWrap
 texreference.addressMode[0]=hipAddressModeClamp;
 texreference.addressMode[1]=hipAddressModeClamp;
 texreference.addressMode[2]=hipAddressModeClamp;

 //bind texture reference with cuda array
 err=hipBindTextureToArray(texreference,cudaarr,channel);
 checkCudaError(err,__FILE__,__LINE__);

 *carray=cudaarr;
}

static void
detach_gpu_from_thread(int card, float *wval, void* carray) {
 hipSetDevice(card);
 hipFree(wval);
 hipArray *cudaarr=(hipArray*)carray;
 //unbind texture reference to free resource
 hipUnbindTexture(texreference);
 hipFreeArray(cudaarr);
}


/* slave thread 2GPU function */
static void *
pipeline_slave_code(void *data)
{
 slave_tdata *td=(slave_tdata*)data;
 gbgdata *gd=(gbgdata*)(td->pline->data);
 int tid=td->tid;
 /* slave barrier */
 th_slave_pipeline tp;
 gb_slave_gdata *tpg;
 int ci;
 int Nt=gd->vis[tid].N;
 if ((tpg=(gb_slave_gdata*)calloc((size_t)Nt,sizeof(gb_slave_gdata)))==0) {
     fprintf(stderr,"%s: %d: no free memory\n",__FILE__,__LINE__);
     exit(1);
 }
 float *wval; /* pointer to wparr w values in GPU */
 while(1) {
  sync_barrier(&(td->pline->gate1)); /* stop at gate 1*/
  if(td->pline->terminate) break; /* if flag is set, break loop */
  sync_barrier(&(td->pline->gate2)); /* stop at gate 2 */
  if (gd->status[tid]==PT_DO_WORK_GRID) {
/************************* work *********************/
  /* update data for slave threads */
  /* wait for slave threads  to finish work */
   sync_barrier(&(tp.gate1)); /* sync at gate 1*/
   for (ci=0; ci<Nt; ci++) {  
    tpg[ci].status=PT_DO_WORK_GRID;
    tpg[ci].vis.flag=gd->vis[tid].flag[ci];
    tpg[ci].vis.u=gd->vis[tid].u[ci];
    tpg[ci].vis.v=gd->vis[tid].v[ci];
    tpg[ci].vis.w=gd->vis[tid].w[ci];
    tpg[ci].vis.wt=gd->vis[tid].wt[ci];
    tpg[ci].vis.xx.x=gd->vis[tid].xx[ci].x;
    tpg[ci].vis.xx.y=gd->vis[tid].xx[ci].y;
    tpg[ci].vis.yy.x=gd->vis[tid].yy[ci].x;
    tpg[ci].vis.yy.y=gd->vis[tid].yy[ci].y;
    tpg[ci].vis.xy.x=gd->vis[tid].xy[ci].x;
    tpg[ci].vis.xy.y=gd->vis[tid].xy[ci].y;
    tpg[ci].vis.yx.x=gd->vis[tid].yx[ci].x;
    tpg[ci].vis.yx.y=gd->vis[tid].yx[ci].y;
   }
   sync_barrier(&(tp.gate2)); /* sync at gate 2*/
   sync_barrier(&(tp.gate1)); /* sync at gate 1*/
   for (ci=0; ci<Nt; ci++) {  
    tpg[ci].status=PT_DO_NOTHING;
   }
   sync_barrier(&(tp.gate2)); /* sync at gate 2*/
/************************* work *********************/
  } else if (gd->status[tid]==PT_DO_AGPU) {
  printf("thread %d : w %dx%d\n",tid,gd->Np,gd->Nw);
#ifndef ONE_GPU
   attach_gpu_to_thread(tid,gd->wkernel,gd->Np,gd->Nw,gd->wparr,&wval,&gd->carray[tid]);
#endif
#ifdef ONE_GPU
   attach_gpu_to_thread(0,gd->wkernel,gd->Np,gd->Nw,gd->wparr,&wval,&gd->carray[tid]);
#endif
   for (ci=0; ci<Nt; ci++) {  
    tpg[ci].status=PT_DO_NOTHING;
#ifdef ONE_GPU
    tpg[ci].card=0;
#endif
#ifndef ONE_GPU
    tpg[ci].card=tid;
#endif
    /* also copy fixed data for gridding */
    tpg[ci].lambda=gd->lambda;
    tpg[ci].uvscale=gd->uvscale;
    tpg[ci].maxW=gd->wparr[gd->Nw]; /* remember max W value here */
    tpg[ci].expW=gd->expW; 
    tpg[ci].deltaU=gd->deltaU;
    tpg[ci].uvgrid=gd->uvgrid;
    tpg[ci].psfgrid=gd->psfgrid;
    tpg[ci].writelock_img=gd->writelock_img;
    tpg[ci].writelock_psf=gd->writelock_psf;
    tpg[ci].Nx=gd->Nx;
    tpg[ci].Ny=gd->Ny;
    tpg[ci].Nw=gd->Nw;
    tpg[ci].wparr=wval;
    tpg[ci].wpsupportX=gd->wpsupportX;
    tpg[ci].wpsupportY=gd->wpsupportY;
    tpg[ci].maxsupport=gd->maxsupport;
    tpg[ci].Np=gd->Np;
    tpg[ci].Nz=gd->Nz;
    tpg[ci].imgmode=gd->imgmode;
   }
   /* debugging: print texture Nx,Ny,Nw values */
   //if (tid==0) {debug_show_texture(0,20,20,1);}
   /* spawn slave threads */
   init_slave_pipeline(&tp,Nt,tpg);
   sync_barrier(&(tp.gate1)); /* sync at gate 1*/
   for (ci=0; ci<Nt; ci++) {  
    tpg[ci].status=PT_DO_AGPU;
   }
   sync_barrier(&(tp.gate2)); /* sync at gate 2*/
   sync_barrier(&(tp.gate1)); /* sync at gate 1*/
   for (ci=0; ci<Nt; ci++) {  
    tpg[ci].status=PT_DO_NOTHING;
   }
   sync_barrier(&(tp.gate2)); /* sync at gate 2*/
  } else if (gd->status[tid]==PT_DO_DGPU) {
   sync_barrier(&(tp.gate1)); /* sync at gate 1*/
   for (ci=0; ci<Nt; ci++) {  
    tpg[ci].status=PT_DO_DGPU;
   }
   sync_barrier(&(tp.gate2)); /* sync at gate 2*/
   sync_barrier(&(tp.gate1)); /* sync at gate 1*/
   for (ci=0; ci<Nt; ci++) {  
    tpg[ci].status=PT_DO_NOTHING;
   }
   sync_barrier(&(tp.gate2)); /* sync at gate 2*/
#ifndef ONE_GPU
   detach_gpu_from_thread(tid,wval,gd->carray[tid]);
#endif
#ifdef ONE_GPU
   detach_gpu_from_thread(0,wval,gd->carray[tid]);
#endif
   /* destroy slave threads */
   destroy_slave_pipeline(&tp,Nt);
  }
 }
 free(tpg);
 return NULL;
}



/* initialize the pipeline
  and start the slaves rolling 
  Ngpu: how many slaves */
void
init_pipeline(th_pipeline *pline, int Ngpu,
     void *data)
{
 pthread_attr_init(&(pline->attr));
 pthread_attr_setdetachstate(&(pline->attr),PTHREAD_CREATE_JOINABLE);

 init_th_barrier(&(pline->gate1),Ngpu+1); /* 3 threads, including master */
 init_th_barrier(&(pline->gate2),Ngpu+1); /* 3 threads, including master */
 pline->terminate=0;
 pline->data=data; /* data should have pointers to t1 and t2 */
 pline->N=Ngpu;

 if ((pline->sd=(slave_tdata**)calloc((size_t)Ngpu,sizeof(slave_tdata*)))==0) {
    fprintf(stderr,"%s: %d: no free memory\n",__FILE__,__LINE__);
    exit(1);
 }
 int ci;
 for (ci=0; ci<Ngpu; ci++) {
  slave_tdata *t0;
  if ((t0=(slave_tdata*)malloc(sizeof(slave_tdata)))==0) {
    fprintf(stderr,"%s: %d: no free memory\n",__FILE__,__LINE__);
    exit(1);
  }
  t0->pline=pline;
  t0->tid=ci;
  pline->sd[ci]=t0;
 }
 if ((pline->slave=(pthread_t*)calloc((size_t)Ngpu,sizeof(pthread_t)))==0) {
    fprintf(stderr,"%s: %d: no free memory\n",__FILE__,__LINE__);
    exit(1);
 }
 for (ci=0; ci<Ngpu; ci++) {
  pthread_create(&(pline->slave[ci]),&(pline->attr),pipeline_slave_code,(void*)pline->sd[ci]);
 }
}

/* destroy the pipeline */
/* need to kill the slaves first */
void
destroy_pipeline(th_pipeline *pline)
{

 pline->terminate=1;
 sync_barrier(&(pline->gate1));
 int ci;
 for (ci=0; ci<pline->N; ci++) {
  pthread_join(pline->slave[ci],NULL);
 }
 destroy_th_barrier(&(pline->gate1));
 destroy_th_barrier(&(pline->gate2));
 pthread_attr_destroy(&(pline->attr));
 for (ci=0; ci<pline->N; ci++) {
  free(pline->sd[ci]);
 }
 free(pline->sd);
 free(pline->slave);
 pline->data=NULL;
}


/* slave thread >2 GPU function */
static void *
pipeline_slave_slave_code(void *data)
{
 slave_slave_tdata *td=(slave_slave_tdata*)data;
 gb_slave_gdata *gd0=(gb_slave_gdata*)(td->pline->data);
 int tid=td->tid;
 gb_slave_gdata *gd=&gd0[tid];
 hipFloatComplex *devimg,*devpsf;
 hipFloatComplex *hostimg,*hostpsf;
 unsigned long int *devoff,*hostoff;

 /* for QUV imaging */
 hipFloatComplex *devimgQ,*devimgU,*devimgV;
 hipFloatComplex *hostimgQ,*hostimgU,*hostimgV;
 /* offsets at NxNy, 2NxNy, 3NxNy */

 hipError_t err;
 int *doutpos;
 /* determine max support to allocate buffer
    not this is full support for +ve and -ve halves */
 int Maxs;
 int nThreads,nBloks;
 int *outpos;
 /* buffer length for host buffers > device buffers */
 int BL=0;
 int bfilled=0;

 while(1) {
  sync_barrier(&(td->pline->gate1)); /* stop at gate 1*/
  if(td->pline->terminate) break; /* if flag is set, break loop */
  sync_barrier(&(td->pline->gate2)); /* stop at gate 2 */
  if (gd->status==PT_DO_WORK_GRID && !gd->vis.flag) {
/************************* work *********************/
   hipSetDevice(gd->card);
   /* find support for this w */
   //printf("nBlocks=%d nThreads=%d\n",nBloks,nThreads);
   cuda_search<<<nBloks, nThreads>>>(doutpos, gd->wparr, gd->vis.w, gd->lambda, gd->Nw+2);
   hipDeviceSynchronize();
   err=hipGetLastError();
   //checkCudaError(err,__FILE__,__LINE__);
   err=hipMemcpy(outpos,doutpos,sizeof(int),hipMemcpyDeviceToHost);
   checkCudaError(err,__FILE__,__LINE__);
   if (*outpos>=gd->Nw) { *outpos=gd->Nw-1; }
   int Mx=gd->wpsupportX[*outpos];
   //printf("card %d kernel %d buff %d Nx=%d Ny=%d Np=%d Nw=%d, plane %d support %d\n",gd->card,tid,Maxs,gd->Nx,gd->Ny,gd->Np,gd->Nw,*outpos-1,Mx);

   if (Mx>0) { /* only when support is finite */
   /* depending on the actual support, adjust dimensions 
      to cover 2D array of Mx x Mx */
    dim3 threadsPerBlock(8, 8);
    dim3 numBlocks((Mx+threadsPerBlock.x-1)/threadsPerBlock.x,
               (Mx+threadsPerBlock.y-1)/threadsPerBlock.y);
//printf("Mx=%d threads=%d,%d blocks=%d,%d\n",Mx,threadsPerBlock.x,threadsPerBlock.y,numBlocks.x,numBlocks.y);
    /* reset offset values to Nx x Ny  to eliminate points off the grid */
    cuda_resetmemory<<<numBlocks,threadsPerBlock>>>(devoff,Mx,gd->Nx,gd->Ny);
    hipDeviceSynchronize();
    int MxM=2*Mx*Mx;
    /* max W is at gd->wparr[gd->Nw] */
    if (gd->imgmode>1) {
     cuda_griddata_iquv<<<numBlocks,threadsPerBlock>>>(devimg,devpsf,devoff,devimgQ,devimgU,devimgV,Mx,gd->vis.u,gd->vis.v,gd->vis.w,gd->vis.wt,make_hipFloatComplex(gd->vis.xx.x,gd->vis.xx.y),make_hipFloatComplex(gd->vis.yy.x,gd->vis.yy.y), make_hipFloatComplex(gd->vis.xy.x,gd->vis.xy.y),make_hipFloatComplex(gd->vis.yx.x,gd->vis.yx.y),gd->maxW,gd->expW,gd->lambda,gd->uvscale,gd->deltaU,gd->Nx,gd->Ny,gd->Nw,gd->Np,gd->Nz);
    } else {
     cuda_griddata<<<numBlocks,threadsPerBlock>>>(devimg,devpsf,devoff,Mx,gd->vis.u,gd->vis.v,gd->vis.w,gd->vis.wt,make_hipFloatComplex(gd->vis.xx.x,gd->vis.xx.y),make_hipFloatComplex(gd->vis.yy.x,gd->vis.yy.y), make_hipFloatComplex(gd->vis.xy.x,gd->vis.xy.y),make_hipFloatComplex(gd->vis.yx.x,gd->vis.yx.y), gd->maxW,gd->expW,gd->lambda,gd->uvscale,gd->deltaU,gd->Nx,gd->Ny,gd->Nw,gd->Np,gd->Nz);
    }
    hipDeviceSynchronize();
    err=hipGetLastError();
    checkCudaError(err,__FILE__,__LINE__);
    /* check if there is enough memory in buffers for copying to host */
    if (bfilled+MxM>=BL) { 
     /* write to output */
     if (gd->imgmode>1) {
      write_buffers_to_output_iquv(bfilled,gd->Nx,gd->Ny,gd->writelock_img,gd->writelock_psf,gd->uvgrid,gd->psfgrid,hostoff,hostimg,hostpsf,hostimgQ,hostimgU,hostimgV,rand()%MAX_GPU);
     } else {
      write_buffers_to_output(bfilled,gd->Nx,gd->Ny,gd->writelock_img,gd->writelock_psf,gd->uvgrid,gd->psfgrid,hostoff,hostimg,hostpsf,rand()%MAX_GPU);
     }
     bfilled=0;
    } 
     err=hipMemcpy(&hostimg[bfilled],devimg,sizeof(hipFloatComplex)*MxM,hipMemcpyDeviceToHost);
     checkCudaError(err,__FILE__,__LINE__);
     err=hipMemcpy(&hostpsf[bfilled],devpsf,sizeof(hipFloatComplex)*MxM,hipMemcpyDeviceToHost);
     checkCudaError(err,__FILE__,__LINE__);
     err=hipMemcpy(&hostoff[bfilled],devoff,sizeof(unsigned long int)*MxM,hipMemcpyDeviceToHost);
     checkCudaError(err,__FILE__,__LINE__);
     if (gd->imgmode>1) {
      err=hipMemcpy(&hostimgQ[bfilled],devimgQ,sizeof(hipFloatComplex)*MxM,hipMemcpyDeviceToHost);
      checkCudaError(err,__FILE__,__LINE__);
      err=hipMemcpy(&hostimgU[bfilled],devimgU,sizeof(hipFloatComplex)*MxM,hipMemcpyDeviceToHost);
      checkCudaError(err,__FILE__,__LINE__);
      err=hipMemcpy(&hostimgV[bfilled],devimgV,sizeof(hipFloatComplex)*MxM,hipMemcpyDeviceToHost);
      checkCudaError(err,__FILE__,__LINE__);
     }
     hipDeviceSynchronize();
     /* advance buffer */
     bfilled+=MxM; 

   }
/************************* work *********************/
  } else if (gd->status==PT_DO_AGPU ) {
   hipSetDevice(gd->card);
   Maxs=gd->maxsupport;
   int MxMs=2*Maxs*Maxs;
   nThreads=16;
   nBloks=(gd->Nw+2+nThreads-1)/nThreads;

   /* for Maxs ~ 512, Maxs*Maxs*2*sizeof(hipFloatComplex) ~= 2MB
      so make buffer size ~ 1 MB by dividing by 2 */
   if (MxMs<DATA_BUF_LEN) {
    BL=DATA_BUF_LEN;
   } else {
    BL=MxMs;
   }
   err=hipMalloc((void**)&devimg,sizeof(hipFloatComplex)*MxMs);
   checkCudaError(err,__FILE__,__LINE__);
   err=hipHostAlloc((void**)&hostimg,sizeof(hipFloatComplex)*BL,hipHostMallocDefault);
   checkCudaError(err,__FILE__,__LINE__);
   err=hipMalloc((void**)&devpsf,sizeof(hipFloatComplex)*MxMs);
   checkCudaError(err,__FILE__,__LINE__);
   err=hipHostAlloc((void**)&hostpsf,sizeof(hipFloatComplex)*BL,hipHostMallocDefault);
   checkCudaError(err,__FILE__,__LINE__);
   err=hipMalloc((void**)&devoff,sizeof(unsigned long int)*MxMs);
   checkCudaError(err,__FILE__,__LINE__);
   err=hipHostAlloc((void**)&hostoff,sizeof(unsigned long int)*BL,hipHostMallocDefault);
   checkCudaError(err,__FILE__,__LINE__);
 
   err=hipMalloc((void**)&doutpos,sizeof(int));
   checkCudaError(err,__FILE__,__LINE__);
   err=hipHostAlloc((void**)&outpos,sizeof(int),hipHostMallocDefault);
   checkCudaError(err,__FILE__,__LINE__);

   if (gd->imgmode>1) {
    /* IQUV imaging */
    err=hipMalloc((void**)&devimgQ,sizeof(hipFloatComplex)*MxMs);
    checkCudaError(err,__FILE__,__LINE__);
    err=hipHostAlloc((void**)&hostimgQ,sizeof(hipFloatComplex)*BL,hipHostMallocDefault);
    checkCudaError(err,__FILE__,__LINE__);
    err=hipMalloc((void**)&devimgU,sizeof(hipFloatComplex)*MxMs);
    checkCudaError(err,__FILE__,__LINE__);
    err=hipHostAlloc((void**)&hostimgU,sizeof(hipFloatComplex)*BL,hipHostMallocDefault);
    checkCudaError(err,__FILE__,__LINE__);
    err=hipMalloc((void**)&devimgV,sizeof(hipFloatComplex)*MxMs);
    checkCudaError(err,__FILE__,__LINE__);
    err=hipHostAlloc((void**)&hostimgV,sizeof(hipFloatComplex)*BL,hipHostMallocDefault);
    checkCudaError(err,__FILE__,__LINE__);
   }

   hipDeviceSynchronize();
  } else if (gd->status==PT_DO_DGPU ) {
   /* write last buffer to output */
   if (gd->imgmode>1) {
    write_buffers_to_output_iquv(bfilled,gd->Nx,gd->Ny,gd->writelock_img,gd->writelock_psf,gd->uvgrid,gd->psfgrid,hostoff,hostimg,hostpsf,hostimgQ,hostimgU,hostimgV,rand()%MAX_GPU);
   } else {
    write_buffers_to_output(bfilled,gd->Nx,gd->Ny,gd->writelock_img,gd->writelock_psf,gd->uvgrid,gd->psfgrid,hostoff,hostimg,hostpsf,rand()%MAX_GPU);
   }
   bfilled=0;
   err=hipFree(doutpos);
   checkCudaError(err,__FILE__,__LINE__);
   err=hipFree(devimg);
   checkCudaError(err,__FILE__,__LINE__);
   err=hipFree(devpsf);
   checkCudaError(err,__FILE__,__LINE__);
   err=hipFree(devoff);
   checkCudaError(err,__FILE__,__LINE__);
   err=hipHostFree(hostimg);
   checkCudaError(err,__FILE__,__LINE__);
   err=hipHostFree(hostpsf);
   checkCudaError(err,__FILE__,__LINE__);
   err=hipHostFree(hostoff);
   checkCudaError(err,__FILE__,__LINE__);
   err=hipHostFree(outpos);
   checkCudaError(err,__FILE__,__LINE__);
   if (gd->imgmode>1) {
    err=hipFree(devimgQ);
    checkCudaError(err,__FILE__,__LINE__);
    err=hipHostFree(hostimgQ);
    checkCudaError(err,__FILE__,__LINE__);
    err=hipFree(devimgU);
    checkCudaError(err,__FILE__,__LINE__);
    err=hipHostFree(hostimgU);
    checkCudaError(err,__FILE__,__LINE__);
    err=hipFree(devimgV);
    checkCudaError(err,__FILE__,__LINE__);
    err=hipHostFree(hostimgV);
    checkCudaError(err,__FILE__,__LINE__);
   }
   hipDeviceSynchronize();
  }
 }
 return NULL;
}



/* initialize the pipeline
  and start the slaves rolling 
N: total slaves
*/
void
init_slave_pipeline(th_slave_pipeline *pline, int N,
     void *data)
{
 if ((pline->sd=(slave_slave_tdata**)malloc(sizeof(slave_slave_tdata*)*N))==0) {
    fprintf(stderr,"no free memory\n");
    exit(1);
 }
 if ((pline->slave=(pthread_t*)malloc(sizeof(pthread_t)*N))==0) {
    fprintf(stderr,"no free memory\n");
    exit(1);
 }

 int ci;
 pthread_attr_init(&(pline->attr));
 pthread_attr_setdetachstate(&(pline->attr),PTHREAD_CREATE_JOINABLE);

 init_th_barrier(&(pline->gate1),N+1); /* N+1 threads, including master */
 init_th_barrier(&(pline->gate2),N+1); /* N+1 threads, including master */
 pline->terminate=0;
 pline->data=data; /* data is an array of data for t1,t2,t3,... */

 for(ci=0; ci<N; ci++) {
  if ((pline->sd[ci]=(slave_slave_tdata*)malloc(sizeof(slave_slave_tdata)))==0) {
    fprintf(stderr,"no free memory\n");
    exit(1);
  }
  pline->sd[ci]->pline=pline;
  pline->sd[ci]->tid=ci;
  pthread_create(&(pline->slave[ci]),&(pline->attr),pipeline_slave_slave_code,(void*)pline->sd[ci]);
 }
}

/* destroy the pipeline */
/* need to kill the slaves first */
void
destroy_slave_pipeline(th_slave_pipeline *pline,int N)
{

 pline->terminate=1;
 sync_barrier(&(pline->gate1));
 int ci;
 for(ci=0; ci<N; ci++) {
  pthread_join(pline->slave[ci],NULL);
  free(pline->sd[ci]);
 }
 destroy_th_barrier(&(pline->gate1));
 destroy_th_barrier(&(pline->gate2));
 pthread_attr_destroy(&(pline->attr));
 pline->data=NULL;
 free(pline->slave);
 free(pline->sd);
}

}
