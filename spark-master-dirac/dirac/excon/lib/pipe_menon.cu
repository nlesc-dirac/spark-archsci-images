#include "hip/hip_runtime.h"
/*
 *
   Copyright (C) 2013 Sarod Yatawatta <sarod@users.sf.net>  
 This program is free software; you can redistribute it and/or modify
 it under the terms of the GNU General Public License as published by
 the Free Software Foundation; either version 2 of the License, or
 (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software
 Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
 $Id$
*/

#include "gridder.h"
#include <glib.h>

//declare texture reference
texture<float,hipTextureType2D,hipReadModeElementType> texreference;

/* sum up all N elements of vector input 
 and save (per block) in output (size > number of blocks) */
__global__ void
plus_reduce_multi(float *input, int N, int blockDim_2, float *output) {
 // Each block loads its elements into shared memory
 extern __shared__ float x[];
 int tid = threadIdx.x;
 int i = blockIdx.x*blockDim.x + threadIdx.x;
 x[tid] = (i<N) ? input[i] : 0.0f; // last block may pad with 0’s
 __syncthreads();
 // Build summation tree over elements, handling case where B is not a power of two.
  int nTotalThreads = blockDim_2; // Total number of threads, rounded up to the next power of two
  while(nTotalThreads > 1) {
   int halfPoint = (nTotalThreads >> 1); // divide by two
    if (tid < halfPoint) {
     int thread2 = tid + halfPoint;
     if (thread2 < blockDim.x) { // Skipping the fictitious threads blockDim.x ... blockDim_2-1
      x[tid] = x[tid]+x[thread2];
     }
    }
    __syncthreads();
    nTotalThreads = halfPoint; // Reducing the binary tree size by two
 }

 /* add back to total */
 if( tid == 0 ) {
  output[blockIdx.x]=x[tid];
 }
}


/* sum up all N elements of vector input 
 NOTE: only 1 block should be used */
__global__ void
plus_reduce(float *input, int N, int blockDim_2, float *total) {
 // Each block loads its elements into shared memory
 extern __shared__ float x[];
 int tid = threadIdx.x;
 int i = blockIdx.x*blockDim.x + threadIdx.x;
 x[tid] = (i<N) ? input[i] : 0.0f; // last block may pad with 0’s
 __syncthreads();
 // Build summation tree over elements, handling case where B is not a power of two.
  int nTotalThreads = blockDim_2; // Total number of threads, rounded up to the next power of two
  while(nTotalThreads > 1) {
   int halfPoint = (nTotalThreads >> 1); // divide by two
    if (tid < halfPoint) {
     int thread2 = tid + halfPoint;
     if (thread2 < blockDim.x) { // Skipping the fictitious threads blockDim.x ... blockDim_2-1
      x[tid] = x[tid]+x[thread2];
     }
    }
    __syncthreads();
    nTotalThreads = halfPoint; // Reducing the binary tree size by two
 }

 /* add back to total */
 if( tid == 0 ) {
  *total=*total+x[tid];
 }
}


__global__ void
kernel_ncpweight(float uf,float vf,float *wtd, float uvscale) {
 __shared__ float a[6];
 __shared__ float b[6];
 __shared__ float c[6];
 __shared__ float x;
 int tid = threadIdx.x;
 if (tid==0) {
  a[0] =0.2589f;
  b[0] =109.4f;
  c[0] =13.09f;
  a[1] =0.6783f;
  b[1] =88.86f;
  c[1] =35.7f;
  a[2] =0.0868f;
  b[2] =212.1f;
  c[2] =10.6f;
  a[3] =-0.5993f;
  b[3] =300.0f;
  c[3] =84.17f;
  a[4] =1.476e+05f;
  b[4] =-4327.0f;
  c[4] =1391.0f;
  a[5] =-5.714f;
  b[5] =13.01f;
  c[5] =185.3f;

  x=sqrtf(uf*uf+vf*vf)/uvscale; /* scale by inverse scale get x in [0,800] */
 }
 __syncthreads();
 if (tid<6) {
  if (x<25.0f||x>900.0f) {
   wtd[tid]=0.0f;
  } else if (x<65.0f) {
   wtd[tid]=2.0517f/(1.0f+expf(-(x-40.0f)*0.333333333f))/6.0f;
  } else if (x>800.f) {
   float x2=(x-800.0f);
   wtd[tid]=0.1832f*expf(-x2*x2*0.001f)/6.0f;
  } else {
   float t=(x-b[tid])/c[tid];
   wtd[tid]=expf(-t*t)*a[tid];
  }
 }
}

__global__ void
kernel_pmconvolution(int N, float *ud, float *vd, float *wtd, float uf, float vf, float *ed) {
  // Each thread saves error into shared memory
  extern __shared__ float ek[];
  int ui=threadIdx.x + blockIdx.x * blockDim.x;
  int tid=threadIdx.x;
  ek[tid]=0.0f;
  if (ui<N) {
   float x=-ud[ui]+uf+0.5f;
   float y=vd[ui]+vf+0.5f;
   /* x,y in [0,1], take absolute value of kernel */
   float tt=fabsf(tex2D(texreference,x,y));
   ek[tid]=wtd[ui]*tt;
//printf("uf,vf %f,%f ud,vd %f,%f x=%f y=%f tt=%f e=%f\n",uf,vf,ud[ui],vd[ui],x,y,tt,ek[tid]);
  }
  __syncthreads();
  // Build summation tree over elements, assuming blockDim.x is power of 2.
  for(int s=blockDim.x/2; s>0; s=s/2) {
    if(tid < s) ek[tid] += ek[tid + s];
   __syncthreads();
  }

  /* copy back the sum to proper location in ed */
  if(tid==0) {
   ed[blockIdx.x]=ek[0];
  }

}

extern "C" {

static void
checkCudaError(hipError_t err, const char *file, int line)
{
    if(!err)
        return;
    fprintf(stderr,"GPU (CUDA): %s %s %d\n", hipGetErrorString(err),file,line);
    exit(EXIT_FAILURE);
}

/* need power of 2 for tree reduction to work */
static int
NearestPowerOf2 (int n){
  if (!n) return n;  //(0 == 2^0)

  int x = 1;
  while(x < n) {
      x <<= 1;
  }
  return x;
}

/* u,v,wt: Nx1 arrays */
static float 
cudakernel_pmconvolution(int card, int N,float *u,float *v,float *wt,float uf,float vf, float uvscale, int convmode) {
  hipError_t err;

  float *ud,*vd,*wtd,*ed;
  float *totald;
  hipSetDevice(card);
  int threadsPerBlock=128;
  int BlocksPerGrid=(N+threadsPerBlock-1)/threadsPerBlock;
  if (BlocksPerGrid==0) { /* catch situation when N=1 */
   BlocksPerGrid=1;
  } 

  err=hipMalloc((void**)&ud, sizeof(float)*(N));
  checkCudaError(err,__FILE__,__LINE__);
  err=hipMalloc((void**)&vd, sizeof(float)*(N));
  checkCudaError(err,__FILE__,__LINE__);
  err=hipMalloc((void**)&wtd, sizeof(float)*((N>6?N:6))); /* note: make sure length>6, because we use it for temp storage */
  checkCudaError(err,__FILE__,__LINE__);
  /* to store sum of each block */
  err=hipMalloc((void**)&ed, sizeof(float)*(BlocksPerGrid));
  checkCudaError(err,__FILE__,__LINE__);

  err=hipMemcpy(ud, (void*)u, sizeof(float)*(N), hipMemcpyHostToDevice);
  checkCudaError(err,__FILE__,__LINE__);
  err=hipMemcpy(vd, (void*)v, sizeof(float)*(N), hipMemcpyHostToDevice);
  checkCudaError(err,__FILE__,__LINE__);
  err=hipMemcpy(wtd, (void*)wt, sizeof(float)*(N), hipMemcpyHostToDevice); 
  checkCudaError(err,__FILE__,__LINE__);

  //printf("card=%d N=%d threads=%d blocks=%d\n",card,N,threadsPerBlock,BlocksPerGrid);
  kernel_pmconvolution<<<BlocksPerGrid,threadsPerBlock,sizeof(float)*threadsPerBlock>>>(N,ud,vd,wtd,uf,vf,ed);
  hipDeviceSynchronize();

  err = hipGetLastError();
  checkCudaError(err,__FILE__,__LINE__);

  err=hipMalloc((void**)&totald, sizeof(float));
  checkCudaError(err,__FILE__,__LINE__);
  hipMemset(totald, 0, sizeof(float));

  /* summation over ed */
  if (BlocksPerGrid<threadsPerBlock) {
    /* one kernel launch is enough */
    plus_reduce<<< 1, BlocksPerGrid, sizeof(float)*BlocksPerGrid>>>(ed, BlocksPerGrid, NearestPowerOf2(BlocksPerGrid), totald);
    hipDeviceSynchronize();
  } else {
    /* multiple kernel launches */
    int L=(BlocksPerGrid+threadsPerBlock-1)/threadsPerBlock;
    /* reuse wtd as temp storage */
    plus_reduce_multi<<< L, threadsPerBlock, sizeof(float)*threadsPerBlock>>>(ed, BlocksPerGrid, NearestPowerOf2(threadsPerBlock), wtd);
    hipDeviceSynchronize();
    plus_reduce<<< 1, L, sizeof(float)*L>>>(wtd, L, NearestPowerOf2(L), totald);
    hipDeviceSynchronize();
  }
  err = hipGetLastError();
  checkCudaError(err,__FILE__,__LINE__);
  float total;
  err=hipMemcpy(&total,totald,sizeof(float),hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  /* now total has (W_k \odot C_k) */
  /* calculate ncp_weight/(W_k \odot C_k): 6 threads for 6 order poly */
  /* reuse wtd as temp storage */
  float ncpwt;
  if (convmode==CONV_MODE_NCP) {
  hipMemset(totald, 0, sizeof(float));
  kernel_ncpweight<<<1,6>>>(uf,vf,wtd,uvscale);
  hipDeviceSynchronize();
  plus_reduce<<< 1, 6, sizeof(float)*6>>>(wtd, 6, NearestPowerOf2(6), totald);
  hipDeviceSynchronize();
  err=hipMemcpy(&ncpwt,totald,sizeof(float),hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  } else {
   ncpwt=1.0f; /* uniform weight */
  }

  err=hipFree(ud);
  checkCudaError(err,__FILE__,__LINE__);
  err=hipFree(vd);
  checkCudaError(err,__FILE__,__LINE__);
  err=hipFree(wtd);
  checkCudaError(err,__FILE__,__LINE__);
  err=hipFree(ed);
  checkCudaError(err,__FILE__,__LINE__);
  err=hipFree(totald);
  checkCudaError(err,__FILE__,__LINE__);
//printf("ncpwt %f conv %f\n",ncpwt,total);
  return ncpwt/(total+1e-12f);
}

/* function to set up a GPU, should be called only once */
static void
attach_gpu_to_thread(int card, float *wkernel, int Np, void **carrayp) {
 hipError_t err;

 hipChannelFormatDesc channel;
 hipArray* carray;
 hipSetDevice(card);
 
 //create channel to describe data type
 channel=hipCreateChannelDesc<float>();

 //allocate device memory for cuda array
 err=hipMallocArray(&carray,&channel,Np,Np);
 checkCudaError(err,__FILE__,__LINE__);

 err=hipMemcpyToArray(carray,0,0,wkernel,sizeof(float)*Np*Np,hipMemcpyHostToDevice);
 checkCudaError(err,__FILE__,__LINE__);


 //all coordinate axes are in [0,1]
 texreference.normalized=true;
 //set texture filter mode property
 //use hipFilterModePoint or hipFilterModeLinear
 texreference.filterMode=hipFilterModeLinear;
 //set texture address mode property
 //use hipAddressModeClamp or hipAddressModeWrap
 texreference.addressMode[0]=hipAddressModeClamp;
 texreference.addressMode[1]=hipAddressModeClamp;

 //bind texture reference with cuda array
 err=hipBindTextureToArray(texreference,carray);
 checkCudaError(err,__FILE__,__LINE__);

 *carrayp=carray;
}

static void
detach_gpu_from_thread(int card, void *carray) {
 hipSetDevice(card);
 hipArray *cudaarr=(hipArray*)carray;
 //unbind texture reference to free resource
 hipUnbindTexture(texreference);
 hipFreeArray(cudaarr);
}


/* slave thread function */
static void *
pipeline_pm_slave_code(void *data)
{
 slave_pmtdata *td=(slave_pmtdata*)data;
 gbpmgdata *gd=(gbpmgdata*)(td->pline->data);
 int tid=td->tid;
 unsigned long int i;
 uvlist *ll;
 int card; /* which GPU */
#ifndef ONE_GPU
 card=tid%2;
#endif
#ifdef ONE_GPU
 card=0;
#endif
 while(1) {
  sync_barrier(&(td->pline->gate1)); /* stop at gate 1*/
  if(td->pline->terminate) break; /* if flag is set, break loop */
  sync_barrier(&(td->pline->gate2)); /* stop at gate 2 */
  if (gd->status[tid]==PT_DO_WORK_GRID) {
/************************* work *********************/
    //printf("thread %d from [%ld,%ld] %ld rows\n",tid,gd->startrow[tid],gd->startrow[tid]+gd->Nrows[tid]-1,gd->Nrows[tid]);
     float *ubuff,*vbuff,*wtbuff;
     int bufsz=1024;
     int bfilled=0;
     if ((ubuff=(float*)malloc(sizeof(float)*(size_t)bufsz))==0) {
        fprintf(stderr,"%s: %d: no free memory\n",__FILE__,__LINE__);
        exit(1);
     }
     if ((vbuff=(float*)malloc(sizeof(float)*(size_t)bufsz))==0) {
        fprintf(stderr,"%s: %d: no free memory\n",__FILE__,__LINE__);
        exit(1);
     }
     if ((wtbuff=(float*)malloc(sizeof(float)*(size_t)bufsz))==0) {
        fprintf(stderr,"%s: %d: no free memory\n",__FILE__,__LINE__);
        exit(1);
     }


     for (i=gd->startrow[tid]; i<gd->startrow[tid]+gd->Nrows[tid]; i++) {
       if (!gd->darr[i].flag) {
//printf("i=%ld u,v %f,%f wold=%f\n",i,gd->darr[i].u,gd->darr[i].v,gd->wold[i]);
         /* calculate right bucket index */
          float tempu = gd->darr[i].u;
          float tempv = gd->darr[i].v;
         /* do all computations using float, till the last moment */
         /* scale to [-0.5,0.5] */
          float uf=tempu*gd->uvscale;
          float vf=tempv*gd->uvscale;
         /* scale to pixel values (never will be > 1/2 image size) */
          float ui=(uf*(float)gd->Nx); /*  width */
          float vi=(vf*(float)gd->Ny); /*  width */
          float poffX=-1.0f;
          float poffY=-1.0f;
        /* use relations  y+vi=(Ny/2-1) and x=ui+Nx/2-1 */
          int x=(int)round(ui+0.5f*gd->Nx+poffX);
          int y=(int)round(-vi+0.5f*gd->Ny+poffY);
         if (x>=0 && x<gd->Nx && y>=0 && y<gd->Ny) {
           int xlow=(x-gd->M>=0?x-gd->M:0);
           int xhigh=(x+gd->M>gd->Nx?gd->Nx:x+gd->M);
           int ylow=(y-gd->M>=0?y-gd->M:0);
           int yhigh=(y+gd->M>gd->Ny?gd->Ny:y+gd->M);
           bfilled=0;
           for (int bx=xlow; bx<xhigh; bx++ )
           for (int by=ylow; by<yhigh; by++ ) {
//printf("U,V %f,%f (u,v) (%f,%f) (x,y) (%d,%d) -> (bx,by) (%d,%d) [xlow,xhigh] (%d,%d) [ylow,yhigh] (%d,%d)\n",tempu,tempv,uf,vf,x,y,bx,by,xlow,xhigh,ylow,yhigh);
            unsigned long int bi=bx*gd->B+by;
            pthread_mutex_lock(gd->writelock_hash);
            ll=(uvlist*)g_hash_table_lookup(gd->ht,&bi);
            pthread_mutex_unlock(gd->writelock_hash);
            if (ll) { /* found neighbour pixel list */
              /* realloc memory if needed */
              if (bfilled+ll->P>bufsz) {
                if((ubuff=(float*)realloc((void*)ubuff,(size_t)(bfilled+ll->P)*sizeof(float)))==0){
                 fprintf(stderr, "%s: %d: no free memory\n", __FILE__,__LINE__);
                 exit(1);
                }
                if((vbuff=(float*)realloc((void*)vbuff,(size_t)(bfilled+ll->P)*sizeof(float)))==0){
                 fprintf(stderr, "%s: %d: no free memory\n", __FILE__,__LINE__);
                 exit(1);
                }
                if((wtbuff=(float*)realloc((void*)wtbuff,(size_t)(bfilled+ll->P)*sizeof(float)))==0){
                 fprintf(stderr, "%s: %d: no free memory\n", __FILE__,__LINE__);
                 exit(1);
                }

                bufsz=bfilled+ll->P;
              }
              /* copy memory */
              memcpy((void*)&ubuff[bfilled],(void*)ll->u,(size_t)(ll->P)*sizeof(float));
              memcpy((void*)&vbuff[bfilled],(void*)ll->v,(size_t)(ll->P)*sizeof(float));
              for (int cw=0; cw<ll->P; cw++) {
                wtbuff[bfilled+cw]=gd->wold[ll->id[cw]];
              }
              bfilled+=ll->P;
            }
           }
           /* now call cuda kernel */
           /* W_k+1 <= (W_k x G_k) / (W_k \odot C_k) */
           /* G_k = 1 for uniform weights, use NCP_WEIGHT function */
           float ratio=cudakernel_pmconvolution(card,bfilled,ubuff,vbuff,wtbuff,uf,vf,gd->uvscale,gd->convmode);
//printf("%ld ratio=%f\n",i,ratio);
//printf("(u,v) (%f,%f) (uf,vf) (%f,%f) (x,y) (%d,%d) [xlow,xhigh] (%d,%d) [ylow,yhigh] (%d,%d)\n",tempu,tempv,uf,vf,x,y,xlow,xhigh,ylow,yhigh);
           gd->wnew[i]=gd->wold[i]*ratio;
//printf("i=%ld old=%f new=%f\n",i,gd->wold[i],gd->wnew[i]);
         }
       }
     }
   
     free(ubuff);
     free(vbuff);
     free(wtbuff);
/************************* work *********************/
  } else if (gd->status[tid]==PT_DO_AGPU) {
  //printf("thread %d : pix %d\n",tid,gd->Np);
   /* FIXME: also copy wparr (w coords) to GPU for searching */
   attach_gpu_to_thread(card,gd->wkernel,gd->Np,&gd->carray[tid]);
  } else if (gd->status[tid]==PT_DO_DGPU) {
   detach_gpu_from_thread(card,gd->carray[tid]);
  }
 }
 return NULL;
}



/* initialize the pipeline
  and start the slaves rolling 
  create 2N slave threads */
void
init_pm_pipeline(th_pmpipeline *pline,
     void *data, int N)
{
 if ((pline->sd=(slave_pmtdata*)malloc(sizeof(slave_pmtdata)*2*N))==0) {
    fprintf(stderr,"no free memory\n");
    exit(1);
 }

 pthread_attr_init(&(pline->attr));
 pthread_attr_setdetachstate(&(pline->attr),PTHREAD_CREATE_JOINABLE);

 init_th_barrier(&(pline->gate1),2*N+1); /* 2N+1 threads, including master */
 init_th_barrier(&(pline->gate2),2*N+1); /* 2N+1 threads, including master */
 pline->terminate=0;
 pline->data=data; /* data should have pointers to t1 and t2 */
 int ci;
 for (ci=0; ci<2*N; ci++) {
  pline->sd[ci].pline=(th_pmpipeline*)pline;
  /* link back t1, t2 to data so they could be freed */
  pline->sd[ci].tid=ci;
 }

 if ((pline->slave=(pthread_t*)malloc(sizeof(pthread_t)*2*N))==0) {
    fprintf(stderr,"no free memory\n");
    exit(1);
 }

 for (ci=0; ci<2*N; ci++) {
  pthread_create(&(pline->slave[ci]),&(pline->attr),pipeline_pm_slave_code,(void*)&pline->sd[ci]);
 }
}

/* destroy the pipeline */
/* need to kill the slaves first */
void
destroy_pm_pipeline(th_pmpipeline *pline, int N)
{
 pline->terminate=1;
 sync_barrier(&(pline->gate1));
 int ci;
 for (ci=0; ci<2*N; ci++) {
  pthread_join(pline->slave[ci],NULL);
 }
 free(pline->slave);
 destroy_th_barrier(&(pline->gate1));
 destroy_th_barrier(&(pline->gate2));
 pthread_attr_destroy(&(pline->attr));
 free(pline->sd);
 pline->data=NULL;
}

}
