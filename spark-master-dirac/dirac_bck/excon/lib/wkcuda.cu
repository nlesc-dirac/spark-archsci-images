#include "hip/hip_runtime.h"
/*
 *
   Copyright (C) 2013 Sarod Yatawatta <sarod@users.sf.net>  
 This program is free software; you can redistribute it and/or modify
 it under the terms of the GNU General Public License as published by
 the Free Software Foundation; either version 2 of the License, or
 (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software
 Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
 $Id$
*/

#include "gridder.h"
#include <hipfft/hipfft.h>
#include <hipblas.h>

__global__ void 
fftshift_1D(hipFloatComplex *u_d, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N) {
    //float a = powf(-1.0f,i&1);
    float a = float(1-2*((i)&1));
    u_d[i].x *= a;
    u_d[i].y *= a;
    }
}
#define IDX2R(i,j,N) (((i)*(N))+(j))
__global__ void 
fftshift_2D(hipFloatComplex *data, int N, int N0) {
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < N && j < N) {
    //float a = powf(-1.0f, (i+j)&1);
    float a = float(1-2*((N0-i+N0-j)&1));
    data[IDX2R(i,j,N)].x *= a;
    data[IDX2R(i,j,N)].y *= a;
    }
}

__global__ void
kernel_lmpswf(hipFloatComplex *din, double *lmgrid, double *denom, float *pswfxy, float w, int Np, int Nz0, int Npad) {
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if (i<Np && j<Np) {
       float phaseterm=float(lmgrid[i*Np+j])*(w);
//printf("i=%d j=%d Np=%d Nz0=%d Npad=%d w=%f phase=%f\n",i,j,Np,Nz0,Npad,w,phaseterm);
       float cosp,sinp;
       sincosf(phaseterm,&sinp,&cosp);
       float invdenom=fdividef(pswfxy[(i+Npad)*Nz0+Npad+j],(float)denom[i*Np+j]);
//printf("i=%d j=%d Np=%d Nz0=%d Npad=%d\n",i,j,Np,Nz0,Npad);
       din[(i+Npad)*Nz0+Npad+j].x=cosp*invdenom;
       din[(i+Npad)*Nz0+Npad+j].y=sinp*invdenom;
//__syncthreads();
    }
}

extern "C" {

static void
checkCudaError(hipError_t err, const char *file, int line) {
    if(!err)
        return;
    fprintf(stderr,"GPU (CUDA): %s %s %d\n", hipGetErrorString(err),file,line);
    exit(EXIT_FAILURE);
}


/*  pswfx: Nz0xNz0
    lmgrid, denom : NpdxNpd
    wkernel: complex float -> cast as float, only copy NpxNp values
*/
void
evaluate_wplane_fft(int Nz0, int Np, int Npd, int Npad, int Npad1, float *pswfxy, double *lmgrid, double *denom, float *wkernel, float *wparr, int Nw, float *peakval) {
  hipError_t err;
  hipfftResult cffterr;
  hipblasStatus_t cbstatus;
  hipblasHandle_t cbhandle;

//printf("Nz0=%d Np=%d Npd=%d Npad=%d Npad1=%d\n",Nz0,Np,Npd,Npad,Npad1);
  float *dpswfxy;
  double *dlmgrid,*ddenom;
  hipFloatComplex *ddin;
  hipfftHandle plan0;
  hipSetDevice(0);
  cbstatus=hipblasCreate(&cbhandle);
  if (cbstatus!=HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr,"%s: %d: CUBLAS create fail\n",__FILE__,__LINE__);
    exit(1);
  }


  cffterr=hipfftPlan2d(&plan0, Nz0, Nz0, HIPFFT_C2C);
  if (cffterr!=HIPFFT_SUCCESS) {
    fprintf(stderr,"%s: %d: CUFFT error\n",__FILE__,__LINE__);
    exit(1);
  }

  err=hipMalloc((void**)&dpswfxy, sizeof(float)*(Nz0*Nz0));
  checkCudaError(err,__FILE__,__LINE__);
  err=hipMalloc((void**)&dlmgrid, sizeof(double)*(Npd*Npd));
  checkCudaError(err,__FILE__,__LINE__);
  err=hipMalloc((void**)&ddenom, sizeof(double)*(Npd*Npd));
  checkCudaError(err,__FILE__,__LINE__);
  err=hipMalloc((void**)&ddin, sizeof(hipFloatComplex)*(Nz0*Nz0));
  checkCudaError(err,__FILE__,__LINE__);

  err=hipMemcpy(dpswfxy,pswfxy,sizeof(float)*Nz0*Nz0,hipMemcpyHostToDevice);
  checkCudaError(err,__FILE__,__LINE__);
  err=hipMemcpy(dlmgrid,lmgrid,sizeof(double)*Npd*Npd,hipMemcpyHostToDevice);
  checkCudaError(err,__FILE__,__LINE__);
  err=hipMemcpy(ddenom,denom,sizeof(double)*Npd*Npd,hipMemcpyHostToDevice);
  checkCudaError(err,__FILE__,__LINE__);

  /* scale by -2pi */
  double alpha=-2.0*M_PI;
  cbstatus=hipblasDscal(cbhandle, Npd*Npd, &alpha, dlmgrid, 1);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((Nz0+threadsPerBlock.x-1)/threadsPerBlock.x,
               (Nz0+threadsPerBlock.y-1)/threadsPerBlock.y);
  dim3 numBlocks1((Npd+threadsPerBlock.x-1)/threadsPerBlock.x,
               (Npd+threadsPerBlock.y-1)/threadsPerBlock.y);

  hipMemset(ddin, 0, sizeof(hipFloatComplex)*Nz0*Nz0);

  err=hipGetLastError(); /* reset errors */
  for (int nw=0; nw<Nw; nw++) {
   kernel_lmpswf<<<numBlocks1,threadsPerBlock>>>(ddin, dlmgrid, ddenom, dpswfxy, wparr[nw], Npd, Nz0, Npad);
   hipDeviceSynchronize();
   err=hipGetLastError();
   checkCudaError(err,__FILE__,__LINE__);


   fftshift_2D<<<numBlocks,threadsPerBlock>>>(ddin, Nz0, Npad+Npd/2);
   hipDeviceSynchronize();
   err=hipGetLastError();
   checkCudaError(err,__FILE__,__LINE__);

   cffterr=hipfftExecC2C(plan0, (hipfftComplex*)ddin, (hipfftComplex *)ddin, HIPFFT_BACKWARD);
   if (cffterr!=HIPFFT_SUCCESS) {
     fprintf(stderr,"%s: %d: CUFFT error\n",__FILE__,__LINE__);
     exit(1);
   }

   fftshift_2D<<<numBlocks,threadsPerBlock>>>(ddin, Nz0, Npad+Npd/2);
   hipDeviceSynchronize();
   err=hipGetLastError();
   checkCudaError(err,__FILE__,__LINE__);

   for (int nrow=Npad+Npad1; nrow<Npad+Npad1+Np; nrow++) {
    /* size is 2 times, because we copy float */
    err=hipMemcpy(&wkernel[nw*Np*Np*2+(nrow-Npad-Npad1)*Np*2],&ddin[nrow*Nz0+Npad+Npad1],sizeof(float)*Np*2,hipMemcpyDeviceToHost);
    checkCudaError(err,__FILE__,__LINE__);
   }
   hipDeviceSynchronize();

   hipMemset(ddin, 0, sizeof(hipFloatComplex)*Nz0*Nz0);
  }

  /* also calculate kernel for w=0, to normalize */
  kernel_lmpswf<<<numBlocks1,threadsPerBlock>>>(ddin, dlmgrid, ddenom, dpswfxy, 0.0f, Npd, Nz0, Npad);
  err=hipGetLastError();
  checkCudaError(err,__FILE__,__LINE__);


  fftshift_2D<<<numBlocks,threadsPerBlock>>>(ddin, Nz0, Npad+Npd/2);
  err=hipGetLastError();
  checkCudaError(err,__FILE__,__LINE__);

  cffterr=hipfftExecC2C(plan0, (hipfftComplex*)ddin, (hipfftComplex *)ddin, HIPFFT_BACKWARD);
  if (cffterr!=HIPFFT_SUCCESS) {
     fprintf(stderr,"%s: %d: CUFFT error\n",__FILE__,__LINE__);
     exit(1);
  }

  hipFloatComplex w0;
  err=hipMemcpy(&w0,&ddin[(Nz0/2)*Nz0+Nz0/2],sizeof(hipFloatComplex),hipMemcpyDeviceToHost);
  checkCudaError(err,__FILE__,__LINE__);

  *peakval=hipCabsf(w0);

  hipfftDestroy(plan0);
  hipFree(dpswfxy);
  hipFree(dlmgrid);
  hipFree(ddenom);
  hipFree(ddin);

  cbstatus=hipblasDestroy(cbhandle);
  if (cbstatus!=HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr,"%s: %d: CUBLAS create fail\n",__FILE__,__LINE__);
    exit(1);
  }

}

}
